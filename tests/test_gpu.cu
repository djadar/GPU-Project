#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
// * Name:       main_gpu.cxx
// * Purpose:    Driver for matrix multiplication on GPU
// * History:    Christophe Picard, Fall 2021
// -----------------------------------------------------------------------------

// includes, system
#include <cmath>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

// Parsing command line options using cxxopts 
// https://github.com/jarro2783/cxxopts.git
//#include "args.hxx"

// Matrix manipulation function
//#include "matrix_utils.h"

// Define different gemm kernel
// #include <gemm_kernel.cuh>
// #include <conv_naive.cuh>
// #include <conv_tiled.cuh>

// Define error checking 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/// ----------------------------------------------------------------------------
/// \fn gpuAssert(hipError_t code, const char *str, int line, bool abort=true)
/// @param[in] code (hipError_t): error code from Cuda
/// @param[in] file (const char *): name of the file containing the error
/// @param[in] line (int): line number containing the error
/// @param[in] abort (bool): force abort on cuda error
/// ----------------------------------------------------------------------------
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    std::cerr << "CUDA error: "<< hipGetErrorString(code)<<" "<< file<<" "<< line<<std::endl;
    if (abort) exit(code);
  }
}

#define REAL float
#define BLOCK_SIZE 1
#define TW 4
#define WK 3


__global__ void
conv_naive( float* out, float* A, float* K, int wK, int wA, int hA)
{
  /* Naive function for calculating convolution between array and 
  * 2D kernel. In future requires tiling the image and the kernel 
  * into smaller pieces before calculating
  *
  * float* out:   output array
  * float* A:     padded input array A
  * float* K:     the filter kernel array
  * int wK:       width of the filter kernel
  * int wA:       width of the input array
  * int hA:       height of the input array
  */

  int pad = wK / 2;
  int w_pad = wA + pad * 2;
  // thread indexing
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float accu = 0.0;
  // Go through each element in the filter kernel
  for(int y=0; y<wK; y++){
    for(int x=0; x<wK; x++){
        // start from (row-k, col-k) position and move through the
        // elements in the kernel
        accu = accu + A[(row + y)*w_pad + col + x] * K[y * wK + x];
      }
  }
  // each thread writes one element to output matrix
  if ((row < hA) && (col < wA)) {
    out[ row * wA + col ] = accu;
    //printf("(%d, %d): %f \n", row, col, accu);
  }
}


__global__ void
conv_tiled( float* out, float* A, float* K, int wK, int wA, int hA)
{
  /*
  Function that calculates the convolution between an array array and filter B.
  The convolution is done in tiles to save global memory access cost.
  Parameters:
    - float* out    Output filtered array
    - float* A      Padded input array
    - float* K      Used filter array
    - int wA        Width of the non-padded input array
    - int hA        Height of the non-padded input array
    - int wK        Width of the kernel

  */

  // kernel to shared memory
  // Faster: some threads load 2 all threads calculate

  // Shared tile
  __shared__ float subTile[TW][TW];
  // TODO: add kernel to shared memory ----------
  
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int i = by * TW + ty;
  int j = bx * TW + tx;

  float accu = 0.0;
  int pad = wK / 2; // Kernel has to always be odd numbered

  // row and col moving forward TW - 2*pad indices per tile
  //   by * 2*pad removed from row to get current tile's row index
  //   bx * 2*pad removed from col to get current tile's col index
  //     -> row index * width + col index results in correct indexing for the subtile
  size_t ind = (i - by * 2*pad) * (wA + 2*pad) + j - bx * 2*pad;
  subTile[ty][tx] = A[ind];

  // Debugging
  // printf("(%d, %d) (%d, %d) Subtile [%d,%d] = A[%d]\n", i, j, bx, by, ty, tx, ind);

  // Sync so all data in subtile is present for calculations. Later there is no need 
  // for another synchronisation because the tiles are independent of each other
  __syncthreads();

  for(int y=0; y<wK; y++){
    for(int x=0; x<wK; x++){
      // start from (row-k, col-k) position and move through the
      // elements in the kernel
      accu += subTile[ty + y][tx + x] * K[y * wK + x];

      // Debugging
      // if ((ty == 0) && (tx == 1))
      //   printf("%d,%d: %f * %f = %f (%d)\n", bx, by, subTile[ty + y][tx + x], K[y * wK + x], subTile[ty + y][tx + x] * K[y * wK + x], y * wK + x);
    }
  }
  // Only the center elements of convolution are needed, skip padded area around calculation
  if ((ty < TW - 2*pad) && (tx < TW - 2*pad)) {
    int outy = (by * (TW - 2*pad) + ty);
    int outx = bx * (TW - 2*pad) + tx;
    // Do not write elements that are outside of output bounds because of tiling
    if ((outx < wA) && (outy < hA)) {
      // Index calculation:
      //   Each row and column of a tile increases the index in that direction by 2*pad
      //   So add multiple of that to each row + the thread row and to each column + the thread column 
      out[outy * wA + outx] = accu;

      // Debugging
      // printf("%d, %d :: %d, %d\n", ty, tx, (by * (TW - 2*pad) + ty) * w, bx * (TW - 2*pad) + tx);
    }
  }
}


__global__ void
conv_tiled_shared( float* out, float* A, float* K, int wA, int hA)
{
  /*
  Function that calculates the convolution between an array array and filter B.
  The convolution is done in tiles to save global memory access cost.
  Parameters:
    - float* out    Output filtered array
    - float* A      Padded input array
    - float* K      Used filter array
    - int wA        Width of the non-padded input array
    - int hA        Height of the non-padded input array
    - int wK        Width of the kernel

  */

  // kernel to shared memory
  // Faster: some threads load 2 all threads calculate

  // Shared tile
  __shared__ float subTile[TW][TW];
  __shared__ float kernel[WK][WK];
  // TODO: add kernel to shared memory ----------
  
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int i = by * TW + ty;
  int j = bx * TW + tx;

  float accu = 0.0;
  int pad = WK / 2; // Kernel has to always be odd numbered

  // row and col moving forward TW - 2*pad indices per tile
  //   by * 2*pad removed from row to get current tile's row index
  //   bx * 2*pad removed from col to get current tile's col index
  //     -> row index * width + col index results in correct indexing for the subtile
  size_t ind = (i - by * 2*pad) * (wA + 2*pad) + j - bx * 2*pad;
  subTile[ty][tx] = A[ind];
  if ((ty < WK) && (tx < WK))
    kernel[ty][tx] = K[ty * WK + tx];

  // Debugging
  // printf("(%d, %d) (%d, %d) Subtile [%d,%d] = A[%d]\n", i, j, bx, by, ty, tx, ind);

  // Sync so all data in subtile is present for calculations. Later there is no need 
  // for another synchronisation because the tiles are independent of each other
  __syncthreads();

  for(int y=0; y<WK; y++){
    for(int x=0; x<WK; x++){
      // start from (row-k, col-k) position and move through the
      // elements in the kernel
      accu += subTile[ty + y][tx + x] * kernel[y][x];

      // Debugging
      // if ((ty == 0) && (tx == 1))
      //   printf("%d,%d: %f * %f = %f (%d)\n", bx, by, subTile[ty + y][tx + x], K[y * wK + x], subTile[ty + y][tx + x] * K[y * wK + x], y * wK + x);
    }
  }
  // Only the center elements of convolution are needed, skip padded area around calculation
  if ((ty < TW - 2*pad) && (tx < TW - 2*pad)) {
    int outy = (by * (TW - 2*pad) + ty);
    int outx = bx * (TW - 2*pad) + tx;
    // Do not write elements that are outside of output bounds because of tiling
    if ((outx < wA) && (outy < hA)) {
      // Index calculation:
      //   Each row and column of a tile increases the index in that direction by 2*pad
      //   So add multiple of that to each row + the thread row and to each column + the thread column 
      out[outy * wA + outx] = accu;

      // Debugging
      // printf("%d, %d :: %d, %d\n", ty, tx, (by * (TW - 2*pad) + ty) * w, bx * (TW - 2*pad) + tx);
    }
  }
}


void sobel_filter(int k, REAL *&A) {
  float v, x_dist, y_dist;
  for (int i = 0; i < k; i++) {
      for (int j = 0; j < k; j++) {
          if (j == floor(k/2)){
              v = 0;
          }
          else {
              y_dist = (i - floor(k/2));
              x_dist = (j - floor(k/2));
              v = x_dist / (x_dist * x_dist + y_dist * y_dist);
          }
          A[i * k + j] = v;
      }
  }
}


void print_array(REAL *&A, int w, int h) {
  std::cout << "[";
  for (int i = 0; i < w*h; i++) {
    std::cout << A[i] << " ";
    if ((i+1)%w ==0){
      std::cout <<"]\n";
      std::cout << "[";
    }
  }
}


///
/// Top level driver
///
int main(int argc, char **argv) {

  std::cout << "[Matrix Multiply Using CUDA] - Starting..." << std::endl;
  int k = 3;
  REAL *kernel = new REAL[k*k];
  sobel_filter(k, kernel);
  print_array(kernel, k, k);
  // Define parser 
  // args::ArgumentParser parser("gemm_cuda", "Matrix Multiply using CUDA");

  // // Set parser value
  // args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
  // args::ValueFlag<int> widthA(parser, "widthA", "Width of matrix A", {"wA"}, 5);
  // args::ValueFlag<int> widthB(parser, "widthB", "Width of matrix B", {"wB"}, 3);
  // args::ValueFlag<int> heightA(parser, "heightA", "Height of matrix A", {"hA"}, 5);
  // args::ValueFlag<int> heightB(parser, "heightB", "Height of matrix B", {"hB"}, 3);
  // //args::ValueFlag<int> blockSize(parser, "blockSize", "Size of blocks", {"sb"}, 32);

  // // Invoke parser
  // try {
  //   parser.ParseCLI(argc, argv);
  // } catch (args::Help) {
  //   std::cout << parser;
  //   return 0;
  // } catch (args::ParseError e) {
  //   std::cerr << e.what() << std::endl;
  //   std::cerr << parser;
  //   return 1;
  // } catch (args::ValidationError e) {
  //   std::cerr << e.what() << std::endl;
  //   std::cerr << parser;
  //   return 1;
  // }

  // Initialize matrix dimensions
  // int WA = args::get(widthA);
  // int WB = args::get(widthB);
  // int HA = args::get(heightA);
  // int HB = args::get(heightB);
  //int BS = args::get(blockSize);



  int WA = 6;
  int WB = 3;
  int HA = 5;
  int HB = 3;
  int WC = WA;
  int HC = HA;

  // Setup CUDA environnement 
  hipError_t error;

  hipDeviceProp_t deviceProp;
  int devID = 0;
  error = hipGetDevice(&devID);

  if (error != hipSuccess) {
    printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
  }
  
  // info
  int dev_count;
  hipGetDeviceCount( &dev_count);
  hipDeviceProp_t dev_prop;
  for (int i = 0; i < dev_count; i++)
    hipGetDeviceProperties( &dev_prop, i);

  error = hipGetDeviceProperties(&deviceProp, devID);

  if (deviceProp.computeMode == hipComputeModeProhibited) {
    std::cerr << "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice() ." <<std::endl;
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess) {
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  } else {
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  // utilities
  hipEvent_t start;
  hipEvent_t stop;
  float msecTotal;

  // allocate host memory for matrices A and B
  unsigned int size_A = (WA + 2) * (HA+2);
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A = (float *)malloc(mem_size_A);
  unsigned int size_B = WB * HB;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B = (float *)malloc(mem_size_B);
  
  // initialize host memory
  //fill_random<REAL>(h_A, WA, HA);
  //fill_random<REAL>(h_B, WB, HB);
  
  float AA [7*8] = {0,0,0,0,0,0,0,0,
                    0,1,1,1,1,1,1,0,
                    0,1,1,1,1,1,1,0,
                    0,1,1,1,1,1,1,0,
                    0,1,1,1,1,1,1,0,
                    0,1,1,1,1,1,1,0,
                    0,0,0,0,0,0,0,0};

  float BB [3*3] = {-0.5, 0.0, 0.5, 
                    -1.0, 0.0, 1.0,
                    -0.5, 0.0, 0.5};
  h_A = AA;
  h_B = BB;


  // allocate device memory
  float *d_A;
  gpuErrchk(hipMalloc((void **)&d_A, mem_size_A));
  float *d_B;
  gpuErrchk(hipMalloc((void **)&d_B, mem_size_B));


  // allocate device memory for result
  unsigned int size_C = WC * HC;



  unsigned int mem_size_C = sizeof(float) * size_C;
  float *d_C;
  gpuErrchk(hipMalloc((void **)&d_C, mem_size_C));

  // allocate host memory for the result
  float *h_C = new float[30];//(float *)malloc(mem_size_C);

  dim3 threads, grid;

  // create and start timer
  hipEventCreate(&start);
  hipEventRecord(start, NULL);
 
  // copy host memory to device
  gpuErrchk(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

  // setup execution parameters
  // threads = dim3(BS, BS);
  // threads=dim3(BLOCK_SIZE, BLOCK_SIZE);
  // grid = dim3(WC / threads.x, HC / threads.y);

  // TODO: Find programmatic values for grids/threads. Currently is calculated only for this example
  // execute the naive kernel
  // threads = dim3(5, 5);
  // grid = dim3(1); 
  // conv_naive<<<grid, threads>>>(d_C, d_A, d_B, 5, 3);

  // execute the tiled kernel
  threads = dim3(TW, TW);
  grid = dim3(3, 3); 
  conv_tiled_shared<<<grid, threads >>>(d_C, d_A, d_B, 6, 5);
  // conv_tiled<<<grid, threads >>>(d_C, d_A, d_B, 3, 6, 5);
  // conv_naive<<<grid, threads >>>(d_C, d_A, d_B, 3, 6, 5);
  gpuErrchk(hipPeekAtLastError());
  
  // copy result from device to host
  gpuErrchk(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));

  
  // stop and destroy timer
  hipEventCreate(&stop);
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msecTotal, start, stop);

  print_array(h_A, 8, 7);
  std::cout << std::endl;
  print_array(h_C, 6, 5);

	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );

  // A and B are static for now
  free(h_C);

  /* Performance computation, results and performance printing ------------ */
  auto flop = 2 * (float)WC * (float)HC * (float)WA;

  std::cout << " == Performances " << std::endl;
  std::cout << "\t Processing time: " << msecTotal << " (ms)"
            << std::endl;
  std::cout << "\t GFLOPS: " << flop / msecTotal / 1e+6 << std::endl;

  return (EXIT_SUCCESS);
}
