#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
// * Name:       main_gpu.cxx
// * Purpose:    Driver for matrix multiplication on GPU
// * History:    Christophe Picard, Fall 2021
// -----------------------------------------------------------------------------

// includes, system
#include <cmath>
#include <ctime>

#include <iostream>
#include <string>

#include <random>
#include <limits>

#include <hip/hip_runtime.h>

// Parsing command line options using cxxopts 
// https://github.com/jarro2783/cxxopts.git
//#include "args.hxx"

// Matrix manipulation function
//#include "matrix_utils.h"

// Define different gemm kernel
// #include <gemm_kernel.cuh>
// #include <conv_naive.cuh>
// #include <conv_tiled.cuh>

// Define error checking 
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/// ----------------------------------------------------------------------------
/// \fn gpuAssert(hipError_t code, const char *str, int line, bool abort=true)
/// @param[in] code (hipError_t): error code from Cuda
/// @param[in] file (const char *): name of the file containing the error
/// @param[in] line (int): line number containing the error
/// @param[in] abort (bool): force abort on cuda error
/// ----------------------------------------------------------------------------
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    std::cerr << "CUDA error: "<< hipGetErrorString(code)<<" "<< file<<" "<< line<<std::endl;
    if (abort) exit(code);
  }
}

#define REAL float
#define BLOCK_SIZE 1
#define TW 4
#define WIDTH_K 3


/// ----------------------------------------------------------------------------
/// \fn void init_mat( int N, T *&A, T *&B)
/// \brief Set matrix coefficients
/// \param A First matrix to initialize 
/// \param B Second matrix to initialize 
/// \param N Size of the matrix
/// ----------------------------------------------------------------------------
template <typename T> 
void fill_random(T *&A, int N, int M) {
  std::mt19937 e(static_cast<unsigned int>(std::time(nullptr)));
  std::uniform_real_distribution<T> f;
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < M; ++j) {
      A[i * M + j] = f(e);
    }
  }
}


void conv(REAL *&out, REAL *&A, REAL *&K, int wK, int wA, int hA){
    float total = 0;
    float elem = 0;
    int w_pad = wA + wK - 1;
    // Go through each pixel in the original array
    for (int r = 0; r < hA; r++) {
        for (int c = 0; c < wA; c++) {
            total = 0;
            // Go through each element in the kernel array
            for (int x = 0; x < wK; x++) {
                for (int y = 0; y < wK; y++) {
                    elem = A[(r + y) * w_pad + c + x];
                    total += elem * K[y * wK + x]; // Add to the total value for the output pixel
                }
            }
            out[r * wA + c] = total;
        }
    }
}

__global__ void
conv_naive( float* out, float* A, float* K, int wK, int wA, int hA)
{
  /* Naive function for calculating convolution between array and 
  * 2D kernel. In future requires tiling the image and the kernel 
  * into smaller pieces before calculating
  *
  * float* out:   output array
  * float* A:     padded input array A
  * float* K:     the filter kernel array
  * int wK:       width of the filter kernel
  * int wA:       width of the input array
  * int hA:       height of the input array
  */

  int pad = wK / 2;
  int w_pad = wA + pad * 2;
  // thread indexing
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float accu = 0.0;
  // Go through each element in the filter kernel
  for(int y=0; y<wK; y++){
    for(int x=0; x<wK; x++){
        // start from (row-k, col-k) position and move through the
        // elements in the kernel
        accu = accu + A[(row + y)*w_pad + col + x] * K[y * wK + x];
      }
  }
  // each thread writes one element to output matrix
  if ((row < hA) && (col < wA)) {
    out[ row * wA + col ] = accu;
    //printf("(%d, %d): %f \n", row, col, accu);
  }
}


__global__ void
conv_tiled( float* out, float* A, float* K, int wK, int wA, int hA)
{
  /*
  Function that calculates the convolution between an array array and filter B.
  The convolution is done in tiles to save global memory access cost.
  Parameters:
    - float* out    Output filtered array
    - float* A      Padded input array
    - float* K      Used filter array
    - int wA        Width of the non-padded input array
    - int hA        Height of the non-padded input array
    - int wK        Width of the kernel

  */

  // kernel to shared memory
  // Faster: some threads load 2 all threads calculate

  // Shared tile
  __shared__ float subTile[TW][TW];
  // TODO: add kernel to shared memory ----------
  
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int i = by * TW + ty;
  int j = bx * TW + tx;

  float accu = 0.0;
  int pad = wK / 2; // Kernel has to always be odd numbered

  // row and col moving forward TW - 2*pad indices per tile
  //   by * 2*pad removed from row to get current tile's row index
  //   bx * 2*pad removed from col to get current tile's col index
  //     -> row index * width + col index results in correct indexing for the subtile
  size_t ind = (i - by * 2*pad) * (wA + 2*pad) + j - bx * 2*pad;
  subTile[ty][tx] = A[ind];

  // Debugging
  // printf("(%d, %d) (%d, %d) Subtile [%d,%d] = A[%d]\n", i, j, bx, by, ty, tx, ind);

  // Sync so all data in subtile is present for calculations. Later there is no need 
  // for another synchronisation because the tiles are independent of each other
  __syncthreads();

  for(int y=0; y<wK; y++){
    for(int x=0; x<wK; x++){
      // start from (row-k, col-k) position and move through the
      // elements in the kernel
      accu += subTile[ty + y][tx + x] * K[y * wK + x];

      // Debugging
      // if ((ty == 0) && (tx == 1))
      //   printf("%d,%d: %f * %f = %f (%d)\n", bx, by, subTile[ty + y][tx + x], K[y * wK + x], subTile[ty + y][tx + x] * K[y * wK + x], y * wK + x);
    }
  }
  // Only the center elements of convolution are needed, skip padded area around calculation
  if ((ty < TW - 2*pad) && (tx < TW - 2*pad)) {
    int outy = (by * (TW - 2*pad) + ty);
    int outx = bx * (TW - 2*pad) + tx;
    // Do not write elements that are outside of output bounds because of tiling
    if ((outx < wA) && (outy < hA)) {
      // Index calculation:
      //   Each row and column of a tile increases the index in that direction by 2*pad
      //   So add multiple of that to each row + the thread row and to each column + the thread column 
      out[outy * wA + outx] = accu;

      // Debugging
      // printf("%d, %d :: %d, %d\n", ty, tx, (by * (TW - 2*pad) + ty) * w, bx * (TW - 2*pad) + tx);
    }
  }
}


__global__ void
conv_tiled_shared( float* out, float* A, float* K, int wA, int hA)
{
  /*
  Function that calculates the convolution between an array array and filter B.
  The convolution is done in tiles to save global memory access cost.
  Parameters:
    - float* out    Output filtered array
    - float* A      Padded input array
    - float* K      Used filter array
    - int wA        Width of the non-padded input array
    - int hA        Height of the non-padded input array
    - int wK        Width of the kernel

  */

  // kernel to shared memory
  // Faster: some threads load 2 all threads calculate

  // Shared tile
  __shared__ float subTile[TW][TW];
  __shared__ float kernel[WIDTH_K][WIDTH_K];
  // TODO: add kernel to shared memory ----------
  
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int i = by * TW + ty;
  int j = bx * TW + tx;

  float accu = 0.0;
  int pad = WIDTH_K / 2; // Kernel has to always be odd numbered

  // row and col moving forward TW - 2*pad indices per tile
  //   by * 2*pad removed from row to get current tile's row index
  //   bx * 2*pad removed from col to get current tile's col index
  //     -> row index * width + col index results in correct indexing for the subtile
  size_t ind = (i - by * 2*pad) * (wA + 2*pad) + j - bx * 2*pad;
  subTile[ty][tx] = A[ind];
  if ((ty < WIDTH_K) && (tx < WIDTH_K))
    kernel[ty][tx] = K[ty * WIDTH_K + tx];

  // Debugging
  // printf("(%d, %d) (%d, %d) Subtile [%d,%d] = A[%d]\n", i, j, bx, by, ty, tx, ind);

  // Sync so all data in subtile is present for calculations. Later there is no need 
  // for another synchronisation because the tiles are independent of each other
  __syncthreads();

  for(int y=0; y<WIDTH_K; y++){
    for(int x=0; x<WIDTH_K; x++){
      // start from (row-k, col-k) position and move through the
      // elements in the kernel
      accu += subTile[ty + y][tx + x] * kernel[y][x];

      // Debugging
      // if ((ty == 0) && (tx == 1))
      //   printf("%d,%d: %f * %f = %f (%d)\n", bx, by, subTile[ty + y][tx + x], K[y * WIDTH_K + x], subTile[ty + y][tx + x] * K[y * WIDTH_K + x], y * wK + x);
    }
  }
  // Only the center elements of convolution are needed, skip padded area around calculation
  if ((ty < TW - 2*pad) && (tx < TW - 2*pad)) {
    int outy = (by * (TW - 2*pad) + ty);
    int outx = bx * (TW - 2*pad) + tx;
    // Do not write elements that are outside of output bounds because of tiling
    if ((outx < wA) && (outy < hA)) {
      // Index calculation:
      //   Each row and column of a tile increases the index in that direction by 2*pad
      //   So add multiple of that to each row + the thread row and to each column + the thread column 
      out[outy * wA + outx] = accu;

      // Debugging
      // printf("%d, %d :: %d, %d\n", ty, tx, (by * (TW - 2*pad) + ty) * w, bx * (TW - 2*pad) + tx);
    }
  }
}


void sobel_filter(int k, REAL *&A) {
  float v, x_dist, y_dist;
  for (int i = 0; i < k; i++) {
      for (int j = 0; j < k; j++) {
          if (j == floor(k/2)){
              v = 0;
          }
          else {
              y_dist = (i - floor(k/2));
              x_dist = (j - floor(k/2));
              v = x_dist / (x_dist * x_dist + y_dist * y_dist);
          }
          A[i * k + j] = v;
      }
  }
}


void print_array(REAL *&A, int w, int h) {
  std::cout << "[";
  for (int i = 0; i < w*h; i++) {
    if (i < w*h - 5)
      continue;
    std::cout << A[i] << " ";
    if ((i+1)%w ==0){
      std::cout <<"]\n";
      std::cout << "[";
    }
  }
  std::cout <<"\n";
}


///
/// Top level driver
///
int main(int argc, char **argv) {

  std::cout << "[Matrix Multiply Using CUDA] - Starting..." << std::endl;
  int k = 3;
  REAL *kernel = new REAL[k*k];
  sobel_filter(k, kernel);
  print_array(kernel, k, k);
  // Define parser 
  // args::ArgumentParser parser("gemm_cuda", "Matrix Multiply using CUDA");

  // // Set parser value
  // args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
  // args::ValueFlag<int> widthA(parser, "widthA", "Width of matrix A", {"wA"}, 5);
  // args::ValueFlag<int> widthB(parser, "widthB", "Width of matrix B", {"wB"}, 3);
  // args::ValueFlag<int> heightA(parser, "heightA", "Height of matrix A", {"hA"}, 5);
  // args::ValueFlag<int> heightB(parser, "heightB", "Height of matrix B", {"hB"}, 3);
  // //args::ValueFlag<int> blockSize(parser, "blockSize", "Size of blocks", {"sb"}, 32);

  // // Invoke parser
  // try {
  //   parser.ParseCLI(argc, argv);
  // } catch (args::Help) {
  //   std::cout << parser;
  //   return 0;
  // } catch (args::ParseError e) {
  //   std::cerr << e.what() << std::endl;
  //   std::cerr << parser;
  //   return 1;
  // } catch (args::ValidationError e) {
  //   std::cerr << e.what() << std::endl;
  //   std::cerr << parser;
  //   return 1;
  // }

  // Initialize matrix dimensions
  // int WA = args::get(widthA);
  // int WB = args::get(widthB);
  // int HA = args::get(heightA);
  // int HB = args::get(heightB);
  //int BS = args::get(blockSize);



  int WA = 5;
  int WK = 3;
  int HA = 5;
  int HK = 3;
  int WC = WA;
  int HC = HA;
  WA = WA + WK - 1;
  HA = HA + WK - 1;

  // Setup CUDA environnement 
  hipError_t error;

  hipDeviceProp_t deviceProp;
  int devID = 0;
  error = hipGetDevice(&devID);

  if (error != hipSuccess) {
    printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
  }
  
  // info
  int dev_count;
  hipGetDeviceCount( &dev_count);
  hipDeviceProp_t dev_prop;
  for (int i = 0; i < dev_count; i++)
    hipGetDeviceProperties( &dev_prop, i);

  error = hipGetDeviceProperties(&deviceProp, devID);

  if (deviceProp.computeMode == hipComputeModeProhibited) {
    std::cerr << "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice() ." <<std::endl;
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess) {
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  } else {
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  // utilities
  hipEvent_t start;
  hipEvent_t stop;
  float msecTotal;

  // allocate host memory for matrices A and B
  unsigned int size_A = (WA) * (HA);
  unsigned int mem_size_A = sizeof(float) * size_A;
  // float *h_A = (float *)malloc(mem_size_A);
  unsigned int size_B = WK * HK;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B = (float *)malloc(mem_size_B);
  
  // initialize host memory
  //fill_random<REAL>(h_A, WA, HA);
  //fill_random<REAL>(h_B, WB, HB);
  
  // float AA [7*8] = {0,0,0,0,0,0,0,0,
  //                   0,1,1,1,1,1,1,0,
  //                   0,1,1,1,1,1,1,0,
  //                   0,1,1,1,1,1,1,0,
  //                   0,1,1,1,1,1,1,0,
  //                   0,1,1,1,1,1,1,0,
  //                   0,0,0,0,0,0,0,0};

  float BB [3*3] = {-0.5, 0.0, 0.5, 
                    -1.0, 0.0, 1.0,
                    -0.5, 0.0, 0.5};
  // h_A = AA;
  h_B = BB;

  float *h_A = new REAL[WA*HA];
  fill_random<REAL>(h_A, WA, HA);


  // allocate device memory
  float *d_A;
  gpuErrchk(hipMalloc((void **)&d_A, mem_size_A));
  float *d_B;
  gpuErrchk(hipMalloc((void **)&d_B, mem_size_B));


  // allocate device memory for result
  unsigned int size_C = WC * HC;

  unsigned int mem_size_C = sizeof(float) * size_C;
  float *d_C;
  gpuErrchk(hipMalloc((void **)&d_C, mem_size_C));

  // allocate host memory for the result
  float *h_C = new float[WC*HC];//(float *)malloc(mem_size_C);

  dim3 threads, grid;

  // create and start timer
  hipEventCreate(&start);
  hipEventRecord(start, NULL);
 
  // copy host memory to device
  gpuErrchk(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

  // setup execution parameters
  // threads = dim3(BS, BS);
  // threads=dim3(BLOCK_SIZE, BLOCK_SIZE);
  // grid = dim3(WC / threads.x, HC / threads.y);

  // TODO: Find programmatic values for grids/threads. Currently is calculated only for this example
  // execute the naive kernel
  // threads = dim3(5, 5);
  // grid = dim3(1); 
  // conv_naive<<<grid, threads>>>(d_C, d_A, d_B, 5, 3);




  print_array(h_A, WA, HA);
  conv(h_C, h_A, h_B, WK, WC, HC);
  print_array(h_C, WC, HC);

  
  threads = dim3(TW, TW);
  grid = dim3(3, 3);

  // conv_naive<<<grid, threads >>>(d_C, d_A, d_B, WK, WC, HC);
  // gpuErrchk(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));
  // print_array(h_C, WC, HC);

  conv_tiled<<<grid, threads >>>(d_C, d_A, d_B, WK, WC, HC);
  gpuErrchk(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));
  print_array(h_C, WC, HC);
  std::cout << "ok";

  // conv_tiled_shared<<<grid, threads >>>(d_C, d_A, d_B, WC, HC);
  // gpuErrchk(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));
  // print_array(h_C, WC, HC);

  gpuErrchk(hipPeekAtLastError());
  
  // stop and destroy timer
  hipEventCreate(&stop);
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msecTotal, start, stop);
  std::cout << "ok2";

  std::cout << std::endl;

	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );

  // A and B are static for now
  free(h_C);

  /* Performance computation, results and performance printing ------------ */
  auto flop = 2 * (float)WC * (float)HC * (float)WA;

  std::cout << " == Performances " << std::endl;
  std::cout << "\t Processing time: " << msecTotal << " (ms)"
            << std::endl;
  std::cout << "\t GFLOPS: " << flop / msecTotal / 1e+6 << std::endl;

  return (EXIT_SUCCESS);
}
