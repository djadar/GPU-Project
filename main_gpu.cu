// -----------------------------------------------------------------------------
// * Name:       main_gpu.cxx
// * Purpose:    Driver for matrix convolutional product on GPU
// -----------------------------------------------------------------------------

// includes, system
#include <cmath>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#include "args.hxx"

// Matrix manipulation function
#include "utils.h"

// Define different convolution kernel
#include <conv_naive.cuh>
#include <conv_shared.cuh>


// Constants
#define REAL float

#define WIDTH_K 3

///
/// Top level driver
///
int main(int argc, char **argv) {

  std::cout << "[Matrix Convolutional Product Using CUDA] - Starting..." << std::endl;

  // Define parser 
  args::ArgumentParser parser("edge_cuda", "Matrix Convolutional Product Using CUDA");

  // Set parser value
  // Set parser value
  args::HelpFlag help(parser, "help", "Display this help menu", {'h', "help"});
  args::ValueFlag<int> widthC(parser, "widthC", "Width of output matrix C", {"WC"},
                              256);
  args::ValueFlag<int> heightC(parser, "heightC", "Height of output matrix C", {"HC"},
                               256);
  args::ValueFlag<int> widthK(parser, "widthB", "Width of kernel matrix K", {"WK"},
                              3);
  args::ValueFlag<int> choice(parser, "choice", "Choose the way of doing the calculation 1 : conv_naive ; 2: conv_tiled ; 3: conv_shared", {"choice"},
                              1);
  // Invoke parser
  try {
    parser.ParseCLI(argc, argv);
  } catch (args::Help) {
    std::cout << parser;
    return 0;
  } catch (args::ParseError e) {
    std::cerr << e.what() << std::endl;
    std::cerr << parser;
    return 1;
  } catch (args::ValidationError e) {
    std::cerr << e.what() << std::endl;
    std::cerr << parser;
    return 1;}
  /*}catch (args::get(choice) !=1 ||args::get(choice) !=2 || args::get(choice) !=3) {
    std::cerr << e.what() << std::endl;
    std::cerr << parser;
    return 1;}*/

  int n = args::get(choice);
  // Initialize matrix dimensions
  int WA, WK, HA, WC, HC;
  WC = args::get(widthC);
  HC = args::get(heightC);
  WK = args::get(widthK);
  WA = WC + WK -1 ;
  HA = HC + WK -1;

  // Setup CUDA environnement 
  hipError_t error;

  hipDeviceProp_t deviceProp;
  int devID = 0;
  error = hipGetDevice(&devID);

  if (error != hipSuccess) {
    printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
  }

  error = hipGetDeviceProperties(&deviceProp, devID);

  if (deviceProp.computeMode == hipComputeModeProhibited) {
    std::cerr << "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice() ." <<std::endl;
    exit(EXIT_SUCCESS);
  }

  if (error != hipSuccess) {
    printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
  } else {
    printf("\t GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  // utilities
  hipEvent_t start;
  hipEvent_t stop;
  float msecTotal;

  // allocate host memory for matrices A and K
  unsigned int mem_size_A = sizeof(float) * WA * HA;
  unsigned int mem_size_K = sizeof(float) * WK * WK;
  unsigned int mem_size_C = sizeof(float) * WC * HC;

  float *h_A = new REAL[WA*HA];
  fill_random<REAL>(h_A, WA, HA);

  REAL *h_K = new REAL[WK*WK];
  sobel_filter(WK, h_K);
  
  // allocate host memory for the result
  float *h_C = new float[WC*HC];
 
  // allocate device memory
  float *d_A;
  hipMalloc((void **)&d_A, mem_size_A);
  float *d_K;
  hipMalloc((void **)&d_K, mem_size_K);
  float *d_C;
  hipMalloc((void **)&d_C, mem_size_C);
  

  // --- Begin calculations ---


  // setup execution parameters
  dim3 threads, grid;
  threads = dim3(TW, TW);
  
  // Print kernel and input
  print_array(h_K, WK, WK);
  print_array(h_A, WA, HA);

  // create and start timer
  hipEventCreate(&start);
  hipEventRecord(start, NULL);

  // copy host memory to device
  hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_K, h_K, mem_size_K, hipMemcpyHostToDevice);

  // Run GPU convolutions
  if (n==1){
      int blocksX = WC / (TW) + 1;
      int blocksY = HC / (TW) + 1;
      grid = dim3(blocksX, blocksY);
      conv_naive<<<grid, threads >>>(d_C, d_A, d_K, WK, WC, HC);
      std::cout << " ================ NAIVE ===================" << std::endl;
    }else if (n==2){
      int blocksX = WC / (TW - WK + 1) + 1;
      int blocksY = HC / (TW - WK + 1) + 1;
      grid = dim3(blocksX, blocksY);
      conv_shared<<<grid, threads >>>(d_C, d_A, d_K, WC, HC);
      std::cout << " ================ SHARED ===================" << std::endl;
    }
  hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

  // stop and destroy timer
  hipEventCreate(&stop);
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);

  

  //Print output
  print_array(h_C, WC, HC);  
  
  // Performance computation, results and performance printing
  hipEventElapsedTime(&msecTotal, start, stop);
  auto flop = 2 * (float)WC * (float)HC * (float)WK * (float)WK;
  int microsecond = msecTotal * 1000;
  std::cout << " == Performances " << std::endl;
  std::cout << "\t Processing time: " << microsecond << " (µs)"
            << std::endl;
  std::cout << "\t GFLOPS: " << flop / microsecond / 1e+3 << std::endl;

	hipFree( d_A );
	hipFree( d_K );
	hipFree( d_C );
  free(h_A);
  free(h_K);
  free(h_C);

  return (EXIT_SUCCESS);
}
